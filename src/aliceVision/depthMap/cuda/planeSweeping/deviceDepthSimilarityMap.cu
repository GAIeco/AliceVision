#include "hip/hip_runtime.h"
// This file is part of the AliceVision project.
// Copyright (c) 2022 AliceVision contributors.
// This Source Code Form is subject to the terms of the Mozilla Public License,
// v. 2.0. If a copy of the MPL was not distributed with this file,
// You can obtain one at https://mozilla.org/MPL/2.0/.

#include "deviceDepthSimilarityMap.hpp"
#include "deviceDepthSimilarityMapKernels.cuh"

#include <aliceVision/depthMap/cuda/host/hostUtils.hpp>

#include <utility>

namespace aliceVision {
namespace depthMap {

__host__ extern void cuda_refineDepthMap(CudaDeviceMemoryPitched<float2, 2>& inout_rcTcDepthSimMap_dmp,
                                         const DeviceCamera& rcDeviceCamera, 
                                         const DeviceCamera& tcDeviceCamera,
                                         const RefineParams& refineParams, 
                                         const ROI& roi,
                                         hipStream_t stream)
{
    // setup block and grid
    const dim3 block(16, 16, 1);
    const dim3 grid(divUp(roi.width(), block.x), divUp(roi.height(), block.y), 1);

    const CudaSize<2>& depthSimMapSize = inout_rcTcDepthSimMap_dmp.getSize();
    CudaDeviceMemoryPitched<float2, 2> bestDepthSimMap_dmp(depthSimMapSize);

    const int halfNSteps = ((refineParams.nDepthsToRefine - 1) / 2) + 1; // default nDepthsToRefine = 31
    const int firstStep = 1 - halfNSteps;

    // find best depth/sim map in depth offset from -(halfNSteps - 1) to (halfNSteps - 1), default from -15 to 15
    for(int step = firstStep; step < halfNSteps; ++step)
    {
        refine_compUpdateYKNCCSimMapPatch_kernel<<<grid, block, 0, stream>>>(
            rcDeviceCamera.getDeviceCamId(),
            tcDeviceCamera.getDeviceCamId(),
            rcDeviceCamera.getTextureObject(), 
            tcDeviceCamera.getTextureObject(),
            inout_rcTcDepthSimMap_dmp.getBuffer(), 
            inout_rcTcDepthSimMap_dmp.getPitch(),
            bestDepthSimMap_dmp.getBuffer(), 
            bestDepthSimMap_dmp.getPitch(), 
            refineParams.wsh, 
            refineParams.gammaC, 
            refineParams.gammaP,
            refineParams.useTcOrRcPixSize, 
            rcDeviceCamera.getWidth(), 
            rcDeviceCamera.getHeight(),
            tcDeviceCamera.getWidth(), 
            tcDeviceCamera.getHeight(),
            step, 
            firstStep,
            roi);
    }

    /* note: filter intermediate refined depth/sim map using bilateral filter or median filter does not improve quality */

    // save the best sim and its direct neighbors sim for interpolation
    CudaDeviceMemoryPitched<float3, 2> lastThreeSimsMap_dmp(depthSimMapSize);

    // set best sim map into lastThreeSimsMap_dmp 
    refine_setLastThreeSimsMap_kernel<<<grid, block, 0, stream>>>(
        lastThreeSimsMap_dmp.getBuffer(), 
        lastThreeSimsMap_dmp.getPitch(),
        bestDepthSimMap_dmp.getBuffer(), 
        bestDepthSimMap_dmp.getPitch(), 
        1, // index 0: (best depth -1), 1: (best depth), 2: (best depth + 1)
        roi); 
  
    {
        // compute similarity of (best depth - 1)
        // note: update best similarity, best depth is unchanged
        refine_compYKNCCSimMapPatch_kernel<<<grid, block, 0, stream>>>(
            rcDeviceCamera.getDeviceCamId(),
            tcDeviceCamera.getDeviceCamId(),
            rcDeviceCamera.getTextureObject(), 
            tcDeviceCamera.getTextureObject(),
            bestDepthSimMap_dmp.getBuffer(), 
            bestDepthSimMap_dmp.getPitch(), 
            refineParams.wsh, 
            refineParams.gammaC, 
            refineParams.gammaP,
            refineParams.useTcOrRcPixSize, 
            rcDeviceCamera.getWidth(), 
            rcDeviceCamera.getHeight(),
            tcDeviceCamera.getWidth(), 
            tcDeviceCamera.getHeight(),
            -1.0f, // best depth - 1
            roi);

        // set similarity of (best depth - 1) into lastThreeSimsMap_dmp
        refine_setLastThreeSimsMap_kernel<<<grid, block, 0, stream>>>(
          lastThreeSimsMap_dmp.getBuffer(), 
          lastThreeSimsMap_dmp.getPitch(),
          bestDepthSimMap_dmp.getBuffer(), 
          bestDepthSimMap_dmp.getPitch(), 
          0, // index 0: (best depth -1), 1: (best depth), 2: (best depth + 1)
          roi); 
    }

    {
        // compute similarity of (best depth + 1)
        // note: update best similarity, best depth is unchanged
        refine_compYKNCCSimMapPatch_kernel<<<grid, block, 0, stream>>>(
            rcDeviceCamera.getDeviceCamId(),
            tcDeviceCamera.getDeviceCamId(),
            rcDeviceCamera.getTextureObject(), 
            tcDeviceCamera.getTextureObject(),
            bestDepthSimMap_dmp.getBuffer(), 
            bestDepthSimMap_dmp.getPitch(), 
            refineParams.wsh, 
            refineParams.gammaC, 
            refineParams.gammaP,
            refineParams.useTcOrRcPixSize, 
            rcDeviceCamera.getWidth(), 
            rcDeviceCamera.getHeight(),
            tcDeviceCamera.getWidth(), 
            tcDeviceCamera.getHeight(),
            +1.0f, // best depth + 1
            roi);

        // set sim of (best depth + 1) into lastThreeSimsMap_dmp
        refine_setLastThreeSimsMap_kernel<<<grid, block, 0, stream>>>(
          lastThreeSimsMap_dmp.getBuffer(), 
          lastThreeSimsMap_dmp.getPitch(),
          bestDepthSimMap_dmp.getBuffer(), 
          bestDepthSimMap_dmp.getPitch(), 
          2, // index 0: (best depth -1), 1: (best depth), 2: (best depth + 1)
          roi); 
    }

    // interpolation from the lastThreeSimsMap_dmp
    refine_interpolateDepthFromThreeSimsMap_kernel<<<grid, block, 0, stream>>>(
      rcDeviceCamera.getDeviceCamId(),
      tcDeviceCamera.getDeviceCamId(),
      lastThreeSimsMap_dmp.getBuffer(), 
      lastThreeSimsMap_dmp.getPitch(), 
      bestDepthSimMap_dmp.getBuffer(), 
      bestDepthSimMap_dmp.getPitch(), 
      refineParams.useTcOrRcPixSize,
      roi);

    inout_rcTcDepthSimMap_dmp.copyFrom(bestDepthSimMap_dmp);
    
    CHECK_CUDA_ERROR();
}

__host__ void cuda_fuseDepthSimMapsGaussianKernelVoting(CudaDeviceMemoryPitched<float2, 2>& out_depthSimMapRefinedFused_dmp,
                                                        const CudaDeviceMemoryPitched<float2, 2>& in_depthSimMapPartSgmUpscale_dmp,
                                                        const std::vector<CudaDeviceMemoryPitched<float2, 2>>& in_depthSimMapPartPerRcTc_dmp,
                                                        const RefineParams& refineParams,
                                                        const ROI& roi, 
                                                        hipStream_t stream)
{
    const float samplesPerPixSize = float(refineParams.nSamplesHalf / ((refineParams.nDepthsToRefine - 1) / 2));
    const float twoTimesSigmaPowerTwo = 2.0f * refineParams.sigma * refineParams.sigma;

    // setup block and grid
    const int blockSize = 16;
    const dim3 block(blockSize, blockSize, 1);
    const dim3 grid(divUp(roi.width(), blockSize), divUp(roi.height(), blockSize), 1);

    const CudaSize<2> roiSize(roi.width(), roi.height());

    assert(roiSize == in_depthSimMapPartPerRcTc_dmp.front().getSize());

    CudaDeviceMemoryPitched<float2, 2> bestGsvSampleMapPart_dmp(roiSize);
    CudaDeviceMemoryPitched<float, 2> gsvSampleMapPart_dmp(roiSize);

    // sliding gaussian window
    for(int sample = -refineParams.nSamplesHalf; sample <= refineParams.nSamplesHalf; ++sample) // default sample range from -150 to 150
    {
        // compute the gaussian window sample 
        for(int tci = 0; tci < in_depthSimMapPartPerRcTc_dmp.size(); ++tci) // number of T cameras
        {
            // sum gaussian window sample score of each RcTc depth/sim map
            fuse_computeGaussianKernelVotingSampleMap_kernel<<<grid, block, 0, stream>>>(
                gsvSampleMapPart_dmp.getBuffer(), 
                gsvSampleMapPart_dmp.getPitch(), 
                in_depthSimMapPartPerRcTc_dmp[tci].getBuffer(), // tc depth/sim map 
                in_depthSimMapPartPerRcTc_dmp[tci].getPitch(),
                in_depthSimMapPartSgmUpscale_dmp.getBuffer(), // sgm depth/pixSize map for middle depth
                in_depthSimMapPartSgmUpscale_dmp.getPitch(),
                tci, // first tc cam id, (re)-initialization
                float(sample),
                samplesPerPixSize, 
                twoTimesSigmaPowerTwo,
                roi);
        }

        // save the sample if it's the best
        fuse_updateBestGaussianKernelVotingSampleMap_kernel<<<grid, block, 0, stream>>>(
            bestGsvSampleMapPart_dmp.getBuffer(), 
            bestGsvSampleMapPart_dmp.getPitch(), 
            gsvSampleMapPart_dmp.getBuffer(),
            gsvSampleMapPart_dmp.getPitch(), 
            sample + refineParams.nSamplesHalf, // first sample, first initialization 
            float(sample),                 
            roi);
    }

    // write the output depth/sim for the best sample
    fuse_computeFusedDepthSimMapFromBestGaussianKernelVotingSampleMap_kernel<<<grid, block, 0, stream>>>(
        out_depthSimMapRefinedFused_dmp.getBuffer(), 
        out_depthSimMapRefinedFused_dmp.getPitch(), 
        bestGsvSampleMapPart_dmp.getBuffer(),
        bestGsvSampleMapPart_dmp.getPitch(), 
        in_depthSimMapPartSgmUpscale_dmp.getBuffer(), // sgm depth/pixSize map for middle depth
        in_depthSimMapPartSgmUpscale_dmp.getPitch(), 
        samplesPerPixSize,
        roi);

    CHECK_CUDA_ERROR();
}

__host__ void cuda_optimizeDepthSimMapGradientDescent(CudaDeviceMemoryPitched<float2, 2>& out_depthSimMapOptimized_dmp,
                                                      const CudaDeviceMemoryPitched<float2, 2>& in_depthSimMapSgmUpscale_dmp,
                                                      const CudaDeviceMemoryPitched<float2, 2>& in_depthSimMapRefinedFused_dmp,
                                                      const DeviceCamera& rcDeviceCamera, 
                                                      const RefineParams& refineParams,
                                                      const ROI& roi,
                                                      hipStream_t stream)
{
    const float samplesPerPixSize = float(refineParams.nSamplesHalf / ((refineParams.nDepthsToRefine - 1) / 2));

    // initialize depth/sim map optimized with SGM depth/sim map
    copy(out_depthSimMapOptimized_dmp, in_depthSimMapSgmUpscale_dmp);
    
    const CudaSize<2> roiSize(roi.width(), roi.height());
    CudaDeviceMemoryPitched<float, 2> optDepthMapPart_dmp(roiSize);
    CudaDeviceMemoryPitched<float, 2> imgVariancePart_dmp(roiSize);

    {
        // setup block and grid
        const dim3 lblock(32, 2, 1);
        const dim3 lgrid(divUp(roi.width(), lblock.x), divUp(roi.height(), lblock.y), 1);

        compute_varLofLABtoW_kernel<<<lgrid, lblock, 0, stream>>>(
            rcDeviceCamera.getTextureObject(), 
            imgVariancePart_dmp.getBuffer(), 
            imgVariancePart_dmp.getPitch(),
            roi);
    }

    CudaTexture<float> imgVarianceTex(imgVariancePart_dmp);

    // setup block and grid
    const int blockSize = 16;
    const dim3 block(blockSize, blockSize, 1);
    const dim3 grid(divUp(roi.width(), blockSize), divUp(roi.height(), blockSize), 1);

    for(int iter = 0; iter < refineParams.nIters; ++iter) // default nb iterations is 100
    {
        // copy depths values from out_depthSimMapOptimized_dmp to optDepthMapPart_dmp
        fuse_getOptDeptMapFromOptDepthSimMap_kernel<<<grid, block, 0, stream>>>(
            optDepthMapPart_dmp.getBuffer(), 
            optDepthMapPart_dmp.getPitch(), 
            out_depthSimMapOptimized_dmp.getBuffer(), // initialized with SGM depth/sim map
            out_depthSimMapOptimized_dmp.getPitch(),
            roi);

        CudaTexture<float> depthTex(optDepthMapPart_dmp);

        // adjust depth/sim by using previously computed depths
        fuse_optimizeDepthSimMap_kernel<<<grid, block, 0, stream>>>(
            rcDeviceCamera.getTextureObject(), 
            rcDeviceCamera.getDeviceCamId(), 
            imgVarianceTex.textureObj,
            depthTex.textureObj, 
            out_depthSimMapOptimized_dmp.getBuffer(), 
            out_depthSimMapOptimized_dmp.getPitch(),
            in_depthSimMapSgmUpscale_dmp.getBuffer(), 
            in_depthSimMapSgmUpscale_dmp.getPitch(),
            in_depthSimMapRefinedFused_dmp.getBuffer(), 
            in_depthSimMapRefinedFused_dmp.getPitch(),
            iter, 
            samplesPerPixSize, 
            roi);
    }

    CHECK_CUDA_ERROR();
}

} // namespace depthMap
} // namespace aliceVision
